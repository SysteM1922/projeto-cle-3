#include "hip/hip_runtime.h"
/**
 * \file main.cu
 *
 * \brief Main module.
 *
 * This module provides the program's logic.
 *
 * \author Guilherme Antunes - 103600
 * \author Pedro Rasinhas - 103541
 */

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <getopt.h>
#include <math.h>

#include "common.h"
#include "hip/hip_runtime.h"

__global__ static void bitonicSort(int *arr, int direction, int N, int K);
__global__ static void validateArray(int *array, int size, int sortType);
__device__ static void sort(int *arr, int sortType, int N);
__device__ static void merge(int *arr, int sortType, int N);

static dim3 getBestGridSize(int iteration);
static dim3 getBestBlockSize(int iteration);
static double get_delta_time(void);

int main(int argc, char **argv)
{
    /* set up the device */
    int dev = 0;

    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    /* parse command line */
    if (argc < 5)
    {
        printf("Usage: %s -f <file> -k <number_of_subsequences>\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    char *fileName = NULL;
    int opt, sortType = 0, k;
    int matrixSize = 1024 * 1024;

    while ((opt = getopt(argc, argv, "k:s:f:h")) != -1)
    {
        switch (opt)
        {
        case 'k': /* k value */
            k = atoi(optarg);
            break;
        case 's': /* sort type */
            sortType = atoi(optarg);
            break;
        case 'f': /* file name */
            fileName = optarg;
            break;
        case 'h': /* help */
            printf("Usage: %s -s <sort_type> -f <file> -k <number_of_subsequences>\n", argv[0]);
            exit(EXIT_SUCCESS);
        default:
            printf("Usage: %s -s <sort_type> -f <file> -k <number_of_subsequences>\n", argv[0]);
            exit(EXIT_FAILURE);
        }
    }

    /* read file */
    int fileSize;
    FILE *file = fopen(fileName, "rb");

    if (file == NULL)
    {
        printf("Error: Could not open file %s\n", fileName);
        exit(EXIT_FAILURE);
    }
    if (fread(&fileSize, sizeof(int), 1, file) != 1)
    {
        printf("Error: Could not read file size\n");
        exit(EXIT_FAILURE);
    }
    printf("File size: %d\n", fileSize);
    int *data = (int *)malloc(matrixSize * sizeof(int));
    if (data == NULL)
    {
        printf("Error: Could not allocate memory\n");
        exit(EXIT_FAILURE);
    }
    if (fread(data, sizeof(int), fileSize, file) != fileSize)
    {
        printf("Error: Could not read file data\n");
        exit(EXIT_FAILURE);
    }
    fclose(file);

    /* reserve memory for the gpu */
    int *d_data;
    CHECK(hipMalloc((void **)&d_data, matrixSize * sizeof(int)));
    CHECK(hipMemcpy(d_data, data, matrixSize * sizeof(int), hipMemcpyHostToDevice));

    int numMerges = log2(k);
    int nrIteractions = log2(fileSize);

    dim3 gridSize = getBestGridSize(numMerges);
    dim3 blockSize = getBestBlockSize(numMerges);

    (void)get_delta_time();

    bitonicSort<<<gridSize, blockSize>>>(d_data, sortType, fileSize, k);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    double dt = get_delta_time();
    printf("GPU time: %f s\n", dt);

    validateArray<<<dim3(1, 1, 1), dim3(1, 1, 1)>>>(d_data, fileSize, sortType);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    free(data);
    CHECK(hipFree(d_data));
    CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}

__global__ static void bitonicSort(int *arr, int sortType, int N, int K)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int idx = blockDim.x * gridDim.x * y + x;

    int size = N / K;

    for (int i = 0; (1 << i) < K + 1; i++)
    {
        if (idx * (1 << i) * N / K >= N)
        {
            return;
        }
        if (i == 0)
        {
            if (idx % 2 == 0)
            {
                sort(arr + idx * (1 << i) * N / K, sortType, size);
            }
            else
            {
                sort(arr + idx * (1 << i) * N / K, !sortType, size);
            }
        }
        else
        {
            if (idx % 2 == 0)
            {
                merge(arr + idx * (1 << i) * N / K, sortType, size);
            }
            else
            {
                merge(arr + idx * (1 << i) * N / K, !sortType, size);
            }
        }
        size <<= 1;
        __syncthreads();
    }
}

__device__ static void sort(int *arr, int sortType, int N)
{
    for (int i = 0; (1 << i) < N; i++)
    {
        for (int j = i + 1; j > 0; j--)
        {
            for (int k = 0; k < N / (1 << j); k++)
            {
                int kj = k * (1 << j);
                if (k * (1 << (j - 1)) / (1 << i) % 2 == sortType)
                {
                    for (int l = 0; l < (1 << (j - 1)); l++)
                    {
                        if (arr[kj + l] > arr[kj + l + (1 << (j - 1))])
                        {
                            int temp = arr[kj + l];
                            arr[kj + l] = arr[kj + l + (1 << (j - 1))];
                            arr[kj + l + (1 << (j - 1))] = temp;
                        }
                    }
                }
                else
                {
                    for (int l = 0; l < (1 << (j - 1)); l++)
                    {
                        if (arr[kj + l] < arr[kj + l + (1 << (j - 1))])
                        {
                            int temp = arr[kj + l];
                            arr[kj + l] = arr[kj + l + (1 << (j - 1))];
                            arr[kj + l + (1 << (j - 1))] = temp;
                        }
                    }
                }
            }
        }
    }
}

__device__ static void merge(int *arr, int sortType, int N)
{
    for (int j = N; j > 0; j >>= 1)
    {
        for (int k = 0; k < N / j; k++)
        {
            int kj = k * j;
            if (k * (j >> 1) / N % 2 == sortType)
            {
                for (int l = 0; l < (j >> 1); l++)
                {
                    if (arr[kj + l] > arr[kj + l + (j >> 1)])
                    {
                        int temp = arr[kj + l];
                        arr[kj + l] = arr[kj + l + (j >> 1)];
                        arr[kj + l + (j >> 1)] = temp;
                    }
                }
            }
            else
            {
                for (int l = 0; l < (j >> 1); l++)
                {
                    if (arr[kj + l] < arr[kj + l + (j >> 1)])
                    {
                        int temp = arr[kj + l];
                        arr[kj + l] = arr[kj + l + (j >> 1)];
                        arr[kj + l + (j >> 1)] = temp;
                    }
                }
            }
        }
    }
}

/**
 *  \brief Function validateArray.
 *
 *  Its role is to validate an integer array.
 *
 *  \param array pointer to the array
 *  \param size array size
 *  \param sortType sort type
 */
__global__ static void validateArray(int *array, int size, int sortType)
{
    int j;
    for (j = 0; j < size - 1; j++)
    {
        if (sortType == (array[j] < array[j + 1]) && array[j] != array[j + 1])
        {
            printf("Error in position %d between element %d and %d\n", j, array[j], array[j + 1]);
            break;
        }
    }
    if (j == (size - 1))
    {
        printf("Everything is OK!\n");
    }
    else
    {
        printf("Something went wrong!\n");
    }
};

static dim3 gridOptions[11] = {
    dim3(1 << 0, 1 << 0, 1 << 0),
    dim3(1 << 0, 1 << 0, 1 << 0),
    dim3(1 << 0, 1 << 0, 1 << 0),
    dim3(1 << 0, 1 << 0, 1 << 0),
    dim3(1 << 0, 1 << 0, 1 << 0),
    dim3(1 << 0, 1 << 0, 1 << 0),
    dim3(1 << 0, 1 << 0, 1 << 0),
    dim3(1 << 0, 1 << 0, 1 << 0),
    dim3(1 << 0, 1 << 0, 1 << 0),
    dim3(1 << 0, 1 << 0, 1 << 0),
    dim3(1 << 0, 1 << 0, 1 << 0),
};

static dim3 blockOptions[11] = {
    dim3(1 << 0, 1 << 0, 1 << 0),
    dim3(1 << 1, 1 << 0, 1 << 0),
    dim3(1 << 2, 1 << 0, 1 << 0),
    dim3(1 << 3, 1 << 0, 1 << 0),
    dim3(1 << 4, 1 << 0, 1 << 0),
    dim3(1 << 5, 1 << 0, 1 << 0),
    dim3(1 << 6, 1 << 0, 1 << 0),
    dim3(1 << 7, 1 << 0, 1 << 0),
    dim3(1 << 8, 1 << 0, 1 << 0),
    dim3(1 << 9, 1 << 0, 1 << 0),
    dim3(1 << 10, 1 << 0, 1 << 0),
};

static dim3 getBestGridSize(int iteration)
{
    return gridOptions[iteration];
};

static dim3 getBestBlockSize(int iteration)
{
    return blockOptions[iteration];
};

static double get_delta_time(void)
{
    static struct timespec t0, t1;

    t0 = t1;
    if (clock_gettime(CLOCK_MONOTONIC, &t1) != 0)
    {
        perror("clock_gettime");
        exit(1);
    }
    return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}